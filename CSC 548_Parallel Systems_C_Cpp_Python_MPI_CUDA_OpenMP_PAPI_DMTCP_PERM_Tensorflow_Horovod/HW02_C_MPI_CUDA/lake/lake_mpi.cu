#include "hip/hip_runtime.h"
/*************************************************************************************************************
 * FILE:            lake_mpi.cu
 *
 * AUTHORS:	        attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *
 * DESCRIPTION:	    Model the surface of a lake, where some pebbles have been thrown onto the surface.
 *                  The energy level at any point on the lake is influenced by
 *                      the energy level on that point in the past,
 *                      and by the current energy levels at neighboring points.
 *                  This program takes into account all 8 neighboring points,
 *                      and parallelizes the simulation by using EXACTLY FOUR compute nodes,
 *                      each using multiple GPU threads.
 *
 * TO RUN:          srun -N4 -n4 -p opteron -x c[53,101,102] --pty /bin/bash
 *                  make -f p3.Makefile lake-mpi
 *                  prun ./lake [lake size] [# pebbles] [duration of simulation in seconds] [# GPU threads]
 *************************************************************************************************************/

// FUNCTION PROTOTYPES
int validate_inputs (int argc, char *argv[]);
int is_number (char sPossibleNumber[]);
void input_validation_error (const char *sMessage);
int allocate_memory ();
int scatter_info ();
int simulate_my_lake_section ();
int start_boundary_exchange ();
int finish_boundary_exchange ();
int update_time (double *nTime, double nTimeStep, double nFinishTime);
int gather_info ();
void report_completion (long int nStartTime_us, long int nEndTime_us);
void shut_down (int nshut_downType);
void initialize_pebbles (double *aPebbleSizes, int nPebbles, int nLakePointsOneAxis);
void initialize_energy (double *aEnergy, double *aPebbleSizes, int nLakePointsOneAxis, int bFullMap);
void print_heatmap (const char *sFilename, double *aEnergy, int bFullMap);
double get_pebble_impact (double nPebbleSize, double nTime);
extern void run_gpu(
    double *aEnergyStepOld,
    double *aEnergyStepCurrent,
    int nLakePointsOneAxis,
    int nNumTaskPointsWithBoundaries,
    double nPointSpacing,
    double nTime,
    int nThreads, int nMyRank,
	int nNumTasks,
	double *P
);


extern void gpu_memory_setup (int nNumTaskPointsWithBoundaries, double *aPebbleSizes);
extern void gpu_memory_free (void);
void report_buffer_double (double *aBuffer, int nNumValues);

// INCLUDES
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <ctype.h>
#include "mpi.h"

// DEFINES
#define N_ROOT_RANK                         0
#define N_TAG_NORTH_EAST                    1
#define N_TAG_NORTH                         2
#define N_TAG_NORTH_WEST                    3
#define N_TAG_WEST                          4
#define N_TAG_SOUTH_WEST                    5
#define N_TAG_SOUTH                         6
#define N_TAG_SOUTH_EAST                    7
#define N_TAG_EAST                          8
#define N_TAG_BOUNDARY_ABOVE_CUR            100
#define N_TAG_BOUNDARY_BELOW_CUR            101
#define N_BAD_INDEX                         -9999
#define N_US_IN_SEC                         1000000
#define XMIN                                0.0
#define XMAX                                1.0
#define YMIN                                0.0
#define YMAX                                1.0
#define MAX_PSZ                             10
#define TSCALE                              1.0
#define VSQR                                0.1
#define B_DEBUG                             0
#define _USE_MATH_DEFINES

// Hack to get around a problem I have in my IDE setup
#ifndef NULL
    #define NULL ((void *) 0)
#endif

// Declare globals

int     nMyRank;
int     nNumTasks;
int     nLakePointsOneAxis;
int     nNumTaskPoints;
int     nNumTaskPointsWithBoundaries;
int     nPebbles;
int     nThreads;

double  nFinishTime;
double  nLakePointSpacing;

double  *aLakePebbleSizes =         NULL;
double  *aLakeEnergyStep0 =         NULL;
double  *aLakeEnergyFinal =         NULL;

double  *aTaskPebbleSizes =         NULL;
double  *aTaskEnergyStepOld =       NULL;
double  *aTaskEnergyStepCurrent =   NULL;

double  *aSendBufferBoundaryAboveCur = NULL;
double  *aSendBufferBoundaryBelowCur = NULL;
double  *aRecvBufferBoundaryAboveCur = NULL;
double  *aRecvBufferBoundaryBelowCur = NULL;

MPI_Request nSendRequestBoundaryAboveCur;
MPI_Request nSendRequestBoundaryBelowCur;
MPI_Request nRecvRequestBoundaryAboveCur;
MPI_Request nRecvRequestBoundaryBelowCur;

/*************************************************************************************************************
 * FUNCTION:        main
 *
 * DESCRIPTION:     Participate the modeling of the surface of a lake,
 *                      where some pebbles have been thrown onto the surface.
 *                  This is one of several nodes participating.
 *                  This and other nodes further parallelize by performing work on multiple GPU threads.
 *
 * ARGUMENTS:       0 -         Lake size (number of points along one axis)
 *                  1 -         Number of pebbles
 *                  2 -         Duration of the simulation in seconds
 *                  3 -         Number of GPU threads along one axis of each thread block
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *************************************************************************************************************/
int main (int argc, char *argv[]) {

    // Declare variables
    int             bOkaySoFar;
    int             nResultMPI;
    long int        nStartTime_us;
    long int        nEndTime_us;
    struct timeval  oStartTime;
    struct timeval  oEndTime;

    // Assume success until we know otherwise
    bOkaySoFar = 1;

    // Initialize MPI
    nResultMPI = MPI_Init(&argc, &argv);
    bOkaySoFar = nResultMPI == MPI_SUCCESS ? 1 : 0;

    /// Get the number of tasks in the communicator
    if (bOkaySoFar == 1) {
        nResultMPI = MPI_Comm_size(MPI_COMM_WORLD, &nNumTasks);
        bOkaySoFar = nResultMPI == MPI_SUCCESS ? 1 : 0;
    }

    // Get my rank in the communicator
    if (bOkaySoFar == 1) {
        nResultMPI = MPI_Comm_rank(MPI_COMM_WORLD, &nMyRank);
        bOkaySoFar = nResultMPI == MPI_SUCCESS ? 1 : 0;
    }

    // Print debug info
    if (bOkaySoFar == 1 && B_DEBUG >= 1) {
        printf("Task rank %d executing main\n", nMyRank);
        fflush(stdout);
    }

    // Validate inputs (after getting comm size, because it matters for validation)
    if (bOkaySoFar == 1) {
        bOkaySoFar = validate_inputs(argc, argv);
    }

    // Save arguments
    if (bOkaySoFar == 1) {
        nLakePointsOneAxis    = atoi(argv[1]);
        nPebbles              = atoi(argv[2]);
        nFinishTime           = (double)atof(argv[3]);
        if (argc >= 5) {
            nThreads = atoi(argv[4]);
        }
        else {
            nThreads = -1;
        }
    }

    // Let the user know what scenario we are running
    if (bOkaySoFar == 1 && nMyRank == N_ROOT_RANK) {
        printf(
            "Running %s on %d nodes with %d threads, with (%d x %d) grid, with %d pebbles, until %f\n",
            argv[0],
            nNumTasks,
            nThreads,
            nLakePointsOneAxis,
            nLakePointsOneAxis,
            nPebbles,
            nFinishTime
        );
    }

    /* Check the wall clock
     * As soon as we know whether we are in charge of measuring time
     *  so that we can include as much as possible in our measurement.
     * Because we want to measure the total time to model the lake
     *  using parallel computation and message passing,
     *  we only want to measure / report this from the root task.
     */
    if (bOkaySoFar == 1 && nMyRank == N_ROOT_RANK) {
        gettimeofday(&oStartTime, NULL);
        nStartTime_us = oStartTime.tv_sec * N_US_IN_SEC + oStartTime.tv_usec;
    }

    // Allocate memory
    if (bOkaySoFar == 1) {
        bOkaySoFar = allocate_memory();
    }

    // Initialize pebbles so we can scatter the info to all tasks
    if (bOkaySoFar == 1 && nMyRank == N_ROOT_RANK) {
        initialize_pebbles(aLakePebbleSizes, nPebbles, nLakePointsOneAxis);
    }

    // Initialize lake energy (given pebble info) so we can print initial heat map
    if (bOkaySoFar == 1 && nMyRank == N_ROOT_RANK) {
        initialize_energy(aLakeEnergyStep0, aLakePebbleSizes, nLakePointsOneAxis, 1);
    }

    // Save lake point spacing before first attempt to print heat map
    if (bOkaySoFar == 1) {
        nLakePointSpacing = (XMAX - XMIN) / nLakePointsOneAxis;
    }

    // Print initial heat map of the entire lake
    if (bOkaySoFar == 1 && nMyRank == N_ROOT_RANK) {
        print_heatmap("lake_i.dat", aLakeEnergyStep0, 1);
    }

    // Barrier to make sure information is scattered when all tasks are ready
    if (bOkaySoFar == 1) {
        nResultMPI = MPI_Barrier(MPI_COMM_WORLD);
        bOkaySoFar = nResultMPI == MPI_SUCCESS ? 1 : 0;
    }

    // Scatter information to all tasks to get started
    if (bOkaySoFar == 1) {
        bOkaySoFar = scatter_info();
    }

    // Initialize energy in my section of the lake (given scattered info) so we can start simulation
    if (bOkaySoFar == 1) {
        initialize_energy(aTaskEnergyStepOld, aTaskPebbleSizes, nLakePointsOneAxis, 0);
        initialize_energy(aTaskEnergyStepCurrent, aTaskPebbleSizes, nLakePointsOneAxis, 0);
    }

    // Simulate the energy changes over time in my section of the lake
    if (bOkaySoFar == 1) {
        bOkaySoFar = simulate_my_lake_section();
    }

    // Barrier to make sure information is gathered when all tasks are ready
    if (bOkaySoFar == 1) {
        nResultMPI = MPI_Barrier(MPI_COMM_WORLD);
        bOkaySoFar = nResultMPI == MPI_SUCCESS ? 1 : 0;
    }

    // Gather information back from all tasks to finish up
    if (bOkaySoFar == 1) {
        bOkaySoFar = gather_info();
    }

    // Print final heat map of the entire lake
    if (bOkaySoFar == 1 && nMyRank == N_ROOT_RANK) {
        print_heatmap("lake_f.dat", aLakeEnergyFinal, 1);
    }

    // If ROOT, measure elapsed time and report completion
    if (bOkaySoFar == 1 && nMyRank == N_ROOT_RANK) {

        // Measure elapsed time
        gettimeofday(&oEndTime, NULL);
        nEndTime_us = oEndTime.tv_sec * N_US_IN_SEC + oEndTime.tv_usec;

        // Report completion to the user
        report_completion(nStartTime_us, nEndTime_us);

    }

    // Shut down
    shut_down(bOkaySoFar);

}

/*************************************************************************************************************
 * FUNCTION:        validate_inputs
 *
 * DESCRIPTION:     Ensure the required command line arguments are present.
 *                  Intended only to be executed by root task
 *
 * ARGUMENTS:       0 -         Lake size (number of points along one axis)
 *                  1 -         Number of pebbles
 *                  2 -         Duration of the simulation in seconds
 *                  3 -         Number of GPU threads along one axis of each thread block
 *                              OPTIONAL: "If GPU threads (nthreads) are specified from the command line,
 *                                          the GPU version should run, otherwise the CPU version"
 *
 * RETURNS:         bSuccess -  1 if inputs look okay, 0 otherwise
 *
 * AUTHOR:          attiffan    Aurora Tiffany-Davis
 *                  wpmoore2    Wade Moore
 *************************************************************************************************************/
int validate_inputs (int argc, char *argv[]) {

    // Declare variables
    int bSuccess;

    // Assume success until we know otherwise
    bSuccess = 1;

    // Check arguments
    if (argc != 5) {
        input_validation_error("Incorrect number of arguments provided");
        bSuccess = 0;
    }
    else if (
        is_number(argv[1]) != 1 ||
        atoi(argv[1]) <= 0
    ) {
        input_validation_error("Lake size must be a positive number");
        bSuccess = 0;
    }
    else if (atoi(argv[1]) % nNumTasks != 0) {
        input_validation_error("Lake size must be evenly divisible by number of tasks");
        bSuccess = 0;
    }
    else if (
        is_number(argv[2]) != 1 ||
        atoi(argv[2]) <= 0
    ) {
        input_validation_error("Number of pebbles must be a positive number");
        bSuccess = 0;
    }
    else if (
        is_number(argv[3]) != 1 ||
        atoi(argv[3]) <= 0
    ) {
        input_validation_error("Duration of simulation must be a positive number");
        bSuccess = 0;
    }
    else if (
        is_number(argv[4]) != 1 ||
        atoi(argv[4]) <= 0
    ) {
        input_validation_error("Number of GPU threads along one axis of each thread block must be a positive number");
        bSuccess = 0;
    }

    // Return
    return bSuccess;
}

/*************************************************************************************************************
 * FUNCTION:        is_number
 *
 * DESCRIPTION:     Checks to see if a char array represents a number
 *
 * ARGUMENTS:       sPossibleNumber - Something we hope actually represents a number
 *
 * RETURN:          1 if number, 0 otherwise
 *
 * SOURCE:          https://stackoverflow.com/questions/29248585/c-checking-command-line-argument-is-integer-or-not
 *
 * AUTHOR:          attiffan    Aurora T. Tiffany-Davis
 *************************************************************************************************************/
int is_number (char sPossibleNumber[]) {

    // Declare variable
    int bNumber = 1;
    int i = 0;

    // Checking for negative numbers
    if (sPossibleNumber[0] == '-') {
        i = 1;
    }
    for (; sPossibleNumber[i] != 0; i++)
    {
        //if (number[i] > '9' || number[i] < '0')
        if (!isdigit(sPossibleNumber[i]) && sPossibleNumber[i] != '.') {
            bNumber = 0;
            break;
        }
    }

    // Return
    return bNumber;

}

/*************************************************************************************************************
 * FUNCTION:        input_validation_error
 *
 * DESCRIPTION:     Complain about an input validation error
 *
 * ARGUMENTS:       sMessage -  A message about the error
 *
 * AUTHOR:          attiffan    Aurora Tiffany-Davis
 *                  wpmoore2    Wade Moore
 *************************************************************************************************************/
void input_validation_error (const char *sMessage) {

    printf("%s\n", sMessage);
    printf("\nUsage: prun ./lake-mpi [lake size] [# pebbles] [duration of simulation in seconds] [# GPU threads]\n");

}

/*************************************************************************************************************
 * FUNCTION:        allocate_memory
 *
 * DESCRIPTION:     Dynamically allocate memory
 *                  (what is needed depends upon task rank)
 *
 * ARGUMENTS:       None
 *
 * RETURN:          bSuccess - 1 if we succeeded, 0 otherwise
 *
 * AUTHOR:          attiffan    Aurora T. Tiffany-Davis
 *************************************************************************************************************/
int allocate_memory () {

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing allocate_memory\n", nMyRank);
        fflush(stdout);
    }

    // Declare variables
    int bSuccess;
    int nNumLakePoints;

    int bExchgAbove = 0;
    int bExchgBelow = 0;

    // Assume success until we know otherwise
    bSuccess = 1;

    /* Figure out who my boundary exchange partners are
     * We assume 4 nodes and the code is NOT expected to work properly with any other number
     * The lake is split up among nodes numbered 0 - 3 as follows:
     *  3
     *  2
     *  1
     *  0
     * If we "exchange above" this means that we are the "above" partner in an exchange
     *  Rank 0 "exchanges below" since it is the "below" partner in a boundary exchange with rank 1
     *  Rank 3 "exchanges above" since it is the "above" partner in a boundary exchange with rank 2
     */
    if (bSuccess == 1) {
        switch (nMyRank) {
            case 0:
                // Send/recv below
                bExchgBelow = 1;
                break;
            case 1:
                // Send/recv above && Send/recv below
                bExchgBelow = 1;
                bExchgAbove = 1;
                break;
            case 2:
                // Send/recv above && Send/recv below
                bExchgBelow = 1;
                bExchgAbove = 1;
                break;
            case 3:
                // Send/recv above
                bExchgAbove = 1;
                break;
            default:
                printf("Task rank %d is not a valid task rank\n", nMyRank);
                bSuccess = 0;
                break;
        }
    }

    // If ROOT, allocate memory for the entire lake
    if (nMyRank == N_ROOT_RANK) {

        // Calculate number of lake points
        nNumLakePoints = nLakePointsOneAxis * nLakePointsOneAxis;

        // Allocate lake-level memory
        if (bSuccess == 1) {
            aLakePebbleSizes = (double*) malloc(sizeof(double) * nNumLakePoints);
            if (aLakePebbleSizes == NULL) {
                printf("Task rank %d could not allocate memory for lake pebble sizes\n", nMyRank);
                bSuccess = 0;
            }
        }
        if (bSuccess == 1) {
            aLakeEnergyStep0 = (double*) malloc(sizeof(double) * nNumLakePoints);
            if (aLakeEnergyStep0 == NULL) {
                printf("Task rank %d could not allocate memory for lake energy (step 0)\n", nMyRank);
                bSuccess = 0;
            }
        }
        if (bSuccess == 1) {
            aLakeEnergyFinal = (double*) malloc(sizeof(double) * nNumLakePoints);
            if (aLakeEnergyFinal == NULL) {
                printf("Task rank %d could not allocate memory for lake energy (final step)\n", nMyRank);
                bSuccess = 0;
            }
        }

    }

    /* Allocate task-level memory
     *  Each node needs enough memory for the energy levels in its own slice of the lake,
     *      plus an extra line at the bottom and the top,
     *      in which to place boundary information received from another node
     *  A node does not need to know about pebbles in the boundaries
     *      touching its slice of the lake,
     *      however pebble array is sized the same as energy array
     *      to make it easier to work with
     */
    nNumTaskPoints = (nLakePointsOneAxis / nNumTasks) * nLakePointsOneAxis;
    nNumTaskPointsWithBoundaries = nNumTaskPoints + (2 * nLakePointsOneAxis);

    if (bSuccess == 1) {
        aTaskPebbleSizes = (double*) malloc(sizeof(double) * nNumTaskPointsWithBoundaries);
        if (aTaskPebbleSizes == NULL) {
            printf("Task rank %d could not allocate memory for task pebble sizes\n", nMyRank);
            bSuccess = 0;
        }
    }
    if (bSuccess == 1) {
        aTaskEnergyStepOld = (double*) malloc(sizeof(double) * nNumTaskPointsWithBoundaries);
        if (aTaskEnergyStepOld == NULL) {
            printf("Task rank %d could not allocate memory for task energy (CPU step old)\n", nMyRank);
            bSuccess = 0;
        }
    }
    if (bSuccess == 1) {
        aTaskEnergyStepCurrent = (double*) malloc(sizeof(double) * nNumTaskPointsWithBoundaries);
        if (aTaskEnergyStepCurrent == NULL) {
            printf("Task rank %d could not allocate memory for task energy (CPU step current)\n", nMyRank);
            bSuccess = 0;
        }
    }

    // Allocate boundary exchange memory

    // ABOVE (we are the "above" partner in a boundary exchange)
    if (bExchgAbove == 1) {

        if (bSuccess == 1) {
          // CURRENT
          // Send buffer
          aSendBufferBoundaryAboveCur = (double*) malloc(sizeof(double) * nLakePointsOneAxis);
          if (aSendBufferBoundaryAboveCur == NULL) {
              printf("Task rank %d could not allocate memory for boundary exchange current above send buffer\n", nMyRank);
              bSuccess = 0;
          }
          // Recv buffer
          aRecvBufferBoundaryAboveCur = (double*) malloc(sizeof(double) * nLakePointsOneAxis);
          if (aRecvBufferBoundaryAboveCur == NULL) {
              printf("Task rank %d could not allocate memory for boundary exchange for current recv buffer\n", nMyRank);
              bSuccess = 0;
          }
       }
    }
    // BELOW (we are the "below" partner in a boundary exchange)
    if (bExchgBelow == 1) {
        // CURRENT
        // Send buffer
        if (bSuccess == 1) {
          aSendBufferBoundaryBelowCur = (double*) malloc(sizeof(double) * nLakePointsOneAxis);
          if (aSendBufferBoundaryBelowCur == NULL) {
            printf("Task rank %d could not allocate memory for boundary exchange for current send buffer\n", nMyRank);
            bSuccess = 0;
          }
        }
        // Recv buffer
        if (bSuccess == 1) {
          aRecvBufferBoundaryBelowCur = (double*) malloc(sizeof(double) * nLakePointsOneAxis);
          if (aRecvBufferBoundaryBelowCur == NULL) {
            printf("Task rank %d could not allocate memory for boundary exchange for current recv buffer\n", nMyRank);
            bSuccess = 0;
          }
        }
    }

    /* Allocate GPU memory
     * GPU needs to know about the pebbles in this node's slice of the lake.
     * GPU does not need to know about pebbles just outside of this slice of the lake,
     *  however pebble array is sized the same as energy arrays,
     *  to make it easier to work with.
     */
    if (bSuccess == 1) {
        gpu_memory_setup(nNumTaskPointsWithBoundaries, aTaskPebbleSizes);
    }

    // Return indication of success / failure
    return bSuccess;

}

/*************************************************************************************************************
 * FUNCTION:        scatter_info
 *
 * DESCRIPTION:     Scatter information out from the root to all tasks to get started
 *
 * ARGUMENTS:       None
 *
 * RETURNS:         bSuccess -      1 if everything seems okay, 0 otherwise
 *
 * AUTHOR:          attiffan        Aurora T. Tiffany-Davis
 *************************************************************************************************************/
int scatter_info () {

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing scatter_info\n", nMyRank);
        fflush(stdout);
    }

    // Declare variables
    int bSuccess;
    int nResultMPI;

    // Assume success until we know otherwise
    bSuccess = 1;

    /* Scatter information out from the root to all tasks to get started
     *  Each task needs to know where the pebbles are in its section of the lake,
     *      and does not need to know about any pebbles in the boundaries
     *      that touch its section of the lake.
     *  However, pebble array is sized the same as energy arrays,
     *      to make it easier to work with.
     *  Energy levels in its section of the lake at time 0 and 1
     *      can then be calculated independently by each task.
     *  Scatter distributes the elements in the order of process rank.
     */
    nResultMPI = MPI_Scatter(
        // Start address of send buffer
        (void *) aLakePebbleSizes,
        // Send count
        nNumTaskPoints,
        // Send type
        MPI_DOUBLE,
        // Start address of receive buffer
        (void *) (aTaskPebbleSizes + nLakePointsOneAxis),
        // Receive count
        nNumTaskPoints,
        // Receive type
        MPI_DOUBLE,
        // Root
        N_ROOT_RANK,
        // Communicator
        MPI_COMM_WORLD
    );
    bSuccess = nResultMPI == MPI_SUCCESS ? 1 : 0;

    // Return
    return bSuccess;

}

/*************************************************************************************************************
 * FUNCTION:        simulate_my_lake_section
 *
 * DESCRIPTION:     Simulate the energy changes over time in my section of the lake
 *
 * ARGUMENTS:       None
 *
 * RETURNS:         bSuccess -      1 if everything seems okay, 0 otherwise
 *
 * AUTHOR:          attiffan        Aurora T. Tiffany-Davis
 *************************************************************************************************************/
int simulate_my_lake_section () {

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing simulate_my_lake_section\n", nMyRank);
        fflush(stdout);
    }

    // Declare variables
    int     bSuccess;
    int     bKeepGoing;
    double  nTime;
    double  nTimeStep;
    char    sFileName[10];

    // Assume success until we know otherwise
    bSuccess = 1;

    // Print initial heat map of my section of the lake
    if (bSuccess == 1) {
        sprintf(sFileName, "lake_i_%d.dat", nMyRank);
        print_heatmap(sFileName, aTaskEnergyStepCurrent, 0);
    }

    // Set initial values for the loop
    nTime =             0.;
    nTimeStep =         nLakePointSpacing / 2.;
    bKeepGoing =        1;

    // Loop through the duration of the simulation
    while (bKeepGoing == 1) {

        // Start boundary exchange (non-blocking)
        if (bSuccess == 1) {
            bSuccess = start_boundary_exchange();
        }

        // Finish boundary exchange (wait for sends and receives to finish)
        if (bSuccess == 1) {
            bSuccess = finish_boundary_exchange();
        }
        /* Simulate the energy changes over one time step in my section of the lake
         *  GPU mode is the only mode that is supported
         *  Each node should communicate boundary information to the appropriate neighbor,
         *  then run the CUDA kernel during a time-step
         *  (one iteration of evolve)
         */

        if (bSuccess == 1) {
            /* Let the GPU update the lake energies for this node's section of the lake
             *  GPU will copy information from CPU to GPU, update values, and copy back to CPU "current" buffer
             *  GPU needs to know how much memory to allocate,
             *      and this must be enough to store
             *      info about this node's slice of the lake,
             *      as well as the boundaries right next to this slice
             */
            run_gpu(
                aTaskEnergyStepOld,
                aTaskEnergyStepCurrent,
                nLakePointsOneAxis,
                nNumTaskPointsWithBoundaries,
                nLakePointSpacing,
                nTime,
                nThreads,
				nMyRank,
				nNumTasks,
				aTaskPebbleSizes
            );

        }

        // Update the simulation time counter
        if (bSuccess == 1) {
            bKeepGoing = update_time(&nTime, nTimeStep, nFinishTime);
        }
    }

    // Free GPU memory
	gpu_memory_free();

    // Print final heat map of my section of the lake
    if (bSuccess == 1) {
        sprintf(sFileName, "lake_f_%d.dat", nMyRank);
        print_heatmap(sFileName, aTaskEnergyStepCurrent, 0);
    }

    // Return
    return bSuccess;

}

/*************************************************************************************************************
 * FUNCTION:        start_boundary_exchange
 *
 * DESCRIPTION:     Start boundary exchanges between tasks for one time step
 *                  Lake is split into exactly 4 quadrants each owned by one compute node
 *                  They are organized in slices.
 *                  Rank 3
 *                      Boundary exchange below with rank 2
 *                  Rank 2
 *                      Boundary exchange above with rank 3
 *                      Boundary exchange below with rank 1
 *                  Rank 1
 *                      Boundary exchange above with rank 2
 *                      Boundary exchange below with rank 0
 *                  Rank 0
 *                      Boundary exchange above with rank 1
 *
 * ARGUMENTS:       None
 *
 * RETURNS:         bSuccess -      1 if everything seems okay, 0 otherwise
 *
 * AUTHOR:          attiffan        Aurora T. Tiffany-Davis
 *************************************************************************************************************/
int start_boundary_exchange () {

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing start_boundary_exchange\n", nMyRank);
        fflush(stdout);
    }

    // Declare variables
    int bSuccess;
    int nResultMPI;
    int bExchgAbove = 0;
    int bExchgBelow = 0;

    // Assume success until we know otherwise
    bSuccess = 1;

    /* Figure out who my boundary exchange partners are
     * We assume 4 nodes and the code is NOT expected to work properly with any other number
     * The lake is split up among nodes numbered 0 - 3 as follows:
     *  3
     *  2
     *  1
     *  0
     * If we "exchange above" this means that we are the "above" partner in an exchange
     *  Rank 0 "exchanges below" since it is the "below" partner in a boundary exchange with rank 1
     *  Rank 3 "exchanges above" since it is the "above" partner in a boundary exchange with rank 2
     */
    if (bSuccess == 1) {
        bExchgBelow = nMyRank < 3 ? 1 : 0;
        bExchgAbove = nMyRank > 0 ? 1 : 0;
    }

    // Perform non-blocking sends from send buffers

    // ABOVE BOUNDARY SEND (we are the "above" partner in a boundary exchange)
    if (bSuccess == 1) {
        if (bExchgAbove == 1) {

            // Fill send boundary buffer with our bottom row skipping our own boundary exchange space
            for (int i = 0; i < nLakePointsOneAxis; i++) {
                aSendBufferBoundaryAboveCur[i] = aTaskEnergyStepCurrent[nLakePointsOneAxis + i];
            }

            if (bSuccess == 1) {
                nResultMPI = MPI_Isend(
                    // Send buffer
                    aSendBufferBoundaryAboveCur,
                    // Send count
                    nLakePointsOneAxis,
                    // Send data type
                    MPI_DOUBLE,
                    // Destination task (one above - zero is bottom)
                    nMyRank - 1,
                    // Tag (named from point of view of SENDER)
                    N_TAG_BOUNDARY_ABOVE_CUR,
                    // Communicator
                    MPI_COMM_WORLD,
                    // Pointer to request
                    &nSendRequestBoundaryAboveCur
                );
                bSuccess = nResultMPI == MPI_SUCCESS ? 1 : 0;
            }
		}
    }

    // BELOW BOUNDARY SEND (we are the "below" partner in a boundary exchange)
	 if (bSuccess == 1) {
        if (bExchgBelow == 1) {
            // Fill send boundary buffer with our top row skipping our own boundary exchange space
            for (int i = 0; i < nLakePointsOneAxis; i++) {
                aSendBufferBoundaryBelowCur[i] = aTaskEnergyStepCurrent[i + nNumTaskPoints];
            }

            if (bSuccess == 1) {
                nResultMPI = MPI_Isend(
                    // Send buffer
                    aSendBufferBoundaryBelowCur,
                    // Send count
                    nLakePointsOneAxis,
                    // Send data type
                    MPI_DOUBLE,
                    // Destination task (one below - zero is bottom)
                    nMyRank + 1,
                    // Tag (named from point of view of SENDER)
                    N_TAG_BOUNDARY_BELOW_CUR,
                    // Communicator
                    MPI_COMM_WORLD,
                    // Pointer to request
                    &nSendRequestBoundaryBelowCur
                );
                bSuccess = nResultMPI == MPI_SUCCESS ? 1 : 0;
            }
		}
	}

	 // ABOVE BOUNDARY RECEIVE (we are the "above" partner in a boundary exchange)
	if (bSuccess == 1) {
		if(bExchgAbove) {
		// Recv boundary (above)
			nResultMPI = MPI_Irecv(
                    // Receive buffer
                    aRecvBufferBoundaryAboveCur,
                    // Receive count
                    nLakePointsOneAxis,
                    // Receive data type
                    MPI_DOUBLE,
                    // Source task (one above - zero is bottom)
                    nMyRank - 1,
                    // Tag (named from point of view of SENDER)
                    N_TAG_BOUNDARY_BELOW_CUR,
                    // Communicator
                    MPI_COMM_WORLD,
                    // Pointer to request
                    &nRecvRequestBoundaryAboveCur
                );
                bSuccess = nResultMPI == MPI_SUCCESS ? 1 : 0;
		}
	}

	// BELOW BOUNDARY RECEIVE (we are the "below" partner in a boundary exchange)
	if (bSuccess == 1) {
		if(bExchgBelow) {
                // Recv boundary (below)
                nResultMPI = MPI_Irecv(
                    // Receive buffer
                    aRecvBufferBoundaryBelowCur,
                    // Receive count
                    nLakePointsOneAxis,
                    // Receive data type
                    MPI_DOUBLE,
                    // Source task (one below - zero is bottom)
                    nMyRank + 1,
                    // Tag (named from point of view of SENDER)
                    N_TAG_BOUNDARY_ABOVE_CUR,
                    // Communicator
                    MPI_COMM_WORLD,
                    // Pointer to request
                    &nRecvRequestBoundaryBelowCur
                );
                bSuccess = nResultMPI == MPI_SUCCESS ? 1 : 0;
            
        }
    }
    // Return
    return bSuccess;

}

/*************************************************************************************************************
 * FUNCTION:        finish_boundary_exchange
 *
 * DESCRIPTION:     Finish boundary exchanges between tasks for one time step
 *
 * ARGUMENTS:       None
 *
 * RETURNS:         bSuccess -      1 if everything seems okay, 0 otherwise
 *
 * AUTHOR:          attiffan        Aurora T. Tiffany-Davis
 *************************************************************************************************************/
int finish_boundary_exchange () {

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing finish_boundary_exchange\n", nMyRank);
        fflush(stdout);
    }

    // Declare variables
    int         bSuccess;
    int         nResultMPI;
    int         nRequestCount;
    int         bExchgAbove;
    int         bExchgBelow;
    int         nBufferOffset;
    int         nRequestIndex;
    MPI_Request aRequests[4];
    MPI_Status  aStatuses[4];

    // Assume success until we know otherwise
    bSuccess = 1;

    /* Figure out who my boundary exchange partners are
     * We assume 4 nodes and the code is NOT expected to work properly with any other number
     * The lake is split up among nodes numbered 0 - 3 as follows:
     *  3
     *  2
     *  1
     *  0
     * If we "exchange above" this means that we are the "above" partner in an exchange
     *  Rank 0 "exchanges below" since it is the "below" partner in a boundary exchange with rank 1
     *  Rank 3 "exchanges above" since it is the "above" partner in a boundary exchange with rank 2
     */
    if (bSuccess == 1) {
        bExchgBelow = nMyRank < 3 ? 1 : 0;
        bExchgAbove = nMyRank > 0 ? 1 : 0;
    }

    // Set up requests
    if (bSuccess == 1) {
        nRequestIndex = 0;
        if (bExchgAbove == 1) {
            aRequests[nRequestIndex] = nSendRequestBoundaryAboveCur;
            nRequestIndex++;
            aRequests[nRequestIndex] = nRecvRequestBoundaryAboveCur;
            nRequestIndex++;
        }
        if (bExchgBelow == 1) {
            aRequests[nRequestIndex] = nSendRequestBoundaryBelowCur;
            nRequestIndex++;
            aRequests[nRequestIndex] = nRecvRequestBoundaryBelowCur;
            nRequestIndex++;
        }
        nRequestCount = nRequestIndex;
    }

    // Wait for all non-blocking sends and receives to complete
    if (bSuccess == 1) {
        nResultMPI = MPI_Waitall(nRequestCount, aRequests, aStatuses);
        bSuccess = nResultMPI == MPI_SUCCESS ? 1 : 0;
    }

    // Copy Receive buffers into task's current energy buffer
    if (bSuccess == 1) {
        /* Use the info that we got as the "above" partner in a boundary exchange
         *  We are copying into an over-sized buffer
         *      meant to hold this boundary exchange info.
         *  The first point we copy over needs to offset - not at all!
         */
        if (bExchgAbove == 1) {
            nBufferOffset = 0;
            for (int i = 0; i < nLakePointsOneAxis; i++) {
                aTaskEnergyStepCurrent[nBufferOffset + i] = aRecvBufferBoundaryAboveCur[i];
            }
        }
        /* Use the info that we got as the "below" partner in a boundary exchange
         *  We are copying into an over-sized buffer
         *      meant to hold this boundary exchange info.
         *  The first point we copy over needs to offset past
         *      the first (below) boundary exchange space,
         *      and also past the space established for this node's slice of the lake.
         */
        if (bExchgBelow == 1) {
            nBufferOffset = nNumTaskPoints + nLakePointsOneAxis;
            for (int i = 0; i < nLakePointsOneAxis; i++) {
                aTaskEnergyStepCurrent[i + nBufferOffset] = aRecvBufferBoundaryBelowCur[i];
            }
        }
    }

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d leaving finish_boundary_exchange\n", nMyRank);
        fflush(stdout);
    }

    // Return
    return bSuccess;

}

/*************************************************************************************************************
 * FUNCTION:        update_time
 *
 * DESCRIPTION:     Update the time counter during the simulation
 *
 * ARGUMENTS:       nTime -         The amount of time that has elapsed in the simulation
 *                  nTimeStep -     The amount of time between one simulation step and the next
 *                  nFinishTime -   The total intended duration of the simulation
 *
 * RETURNS:         bKeepGoing -    1 if we should keep going, 0 otherwise
 *
 * AUTHOR:          attiffan        Aurora T. Tiffany-Davis
 *************************************************************************************************************/
int update_time (double *nTime, double nTimeStep, double nFinishTime) {

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing update_time\n", nMyRank);
        fflush(stdout);
    }

    // Declare variables
    int bKeepGoing;

    // Update the time counter
    if((*nTime) + nTimeStep > nFinishTime) {
        bKeepGoing = 0;
    }
    else {
        (*nTime) = (*nTime) + nTimeStep;
        bKeepGoing = 1;
    }

    // Return
    return bKeepGoing;

}

/*************************************************************************************************************
 * FUNCTION:        gather_info
 *
 * DESCRIPTION:     Gather information back to the root from all tasks to finish up
 *
 * ARGUMENTS:       None
 *
 * RETURNS:         bSuccess -      1 if everything seems okay, 0 otherwise
 *
 * AUTHOR:          attiffan        Aurora T. Tiffany-Davis
 *************************************************************************************************************/
int gather_info () {

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing gather_info\n", nMyRank);
        fflush(stdout);
    }

    // Declare variables
    int bSuccess;
    int nResultMPI;

    // Assume success until we know otherwise
    bSuccess = 1;

    /* Gather information back to the root from all tasks to finish up
     *  The root needs to know the final energy levels in every point of the lake.
     *  Each task is responsible for a separate section of the lake,
     *      and we gather back only each node's own section
     *  We are careful to exclude the extra memory space allocated on each node
     *      to store boundary exchange values,
     *      so we offset the pointer to our energy array,
     *      doing the arithmetic before casting to the required void pointer.
     *  Elements are ordered in the receive buffer
     *      by the rank of the process from which they were received.
     *  Only the root process needs to have a valid receive buffer.
     */
    nResultMPI = MPI_Gather(
        // Start address of send buffer
        (void *) (aTaskEnergyStepCurrent + nLakePointsOneAxis),
        // Send count
        nNumTaskPoints,
        // Send type
        MPI_DOUBLE,
        // Start address of receive buffer
        (void *) aLakeEnergyFinal,
        // Receive count
        nNumTaskPoints,
        // Receive type
        MPI_DOUBLE,
        // Root
        N_ROOT_RANK,
        // Communicator
        MPI_COMM_WORLD
    );
    bSuccess = nResultMPI == MPI_SUCCESS ? 1 : 0;

    // Return
    return bSuccess;

}

/*************************************************************************************************************
 * FUNCTION:        report_completion
 *
 * DESCRIPTION:     Report the completion of the entire job
 *                  Intention is to call this only from the root rank
 *
 * ARGUMENTS:       nStartTime_ms - Start time for the entire job, in microseconds
 *                  nEndTime_ms -   End time for the entire job, in microseconds
 *
 * RETURNS:         None
 *
 * AUTHOR:          attiffan        Aurora T. Tiffany-Davis
 *************************************************************************************************************/
void report_completion (long int nStartTime_us, long int nEndTime_us) {

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing report_completion\n", nMyRank);
        fflush(stdout);
    }

    // Report results
    printf("\nLake size (points along one axis): %d\n", nLakePointsOneAxis);
    printf("Number of pebbles: %d\n", nPebbles);
    printf("Duration of simulation (s): %f\n", nFinishTime);
    printf("Number of GPU threads along one axis of each thread block: %d\n", nThreads);
    printf("Total job run time (s): %f\n\n", ((double) (nEndTime_us - nStartTime_us) / N_US_IN_SEC));

}

/*************************************************************************************************************
 * FUNCTION:        shut_down
 *
 * DESCRIPTION:     Shut down (free dynamically allocated memory, finalize MPI)
 *
 * ARGUMENTS:       bOkaySoFar - 1 if everything seems okay, 0 otherwise
 *
 * AUTHOR:          attiffan    Aurora T. Tiffany-Davis
 *************************************************************************************************************/
void shut_down (int bOkaySoFar) {

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing shut_down\n", nMyRank);
        fflush(stdout);
    }

    // Declare variables
    int bSuccess;
    int nResultMPI;

    // Wait until all tasks get here (helps with debug)
    nResultMPI = MPI_Barrier(MPI_COMM_WORLD);
    bSuccess = (bOkaySoFar == 1 && nResultMPI == MPI_SUCCESS) ? 1 : 0;

    // Free lake-level allocated memory
    if (aLakePebbleSizes != NULL) {
        free(aLakePebbleSizes);
    }
    if (aLakeEnergyStep0 != NULL) {
        free(aLakeEnergyStep0);
    }
    if (aLakeEnergyFinal != NULL) {
        free(aLakeEnergyFinal);
    }

    // Free task-level allocated memory
    if (aTaskPebbleSizes != NULL) {
        free(aTaskPebbleSizes);
    }
    if (aTaskEnergyStepOld != NULL) {
        free(aTaskEnergyStepOld);
    }
    if (aTaskEnergyStepCurrent != NULL) {
        free(aTaskEnergyStepCurrent);
    }

    // Free boundary exchange buffers
    if (aSendBufferBoundaryAboveCur != NULL) {
        free(aSendBufferBoundaryAboveCur);
    }
    if (aSendBufferBoundaryBelowCur != NULL) {
        free(aSendBufferBoundaryBelowCur);
    }
    if (aRecvBufferBoundaryAboveCur != NULL) {
        free(aRecvBufferBoundaryAboveCur);
    }
    if (aRecvBufferBoundaryBelowCur != NULL) {
        free(aRecvBufferBoundaryBelowCur);
    }

    // Free GPU memory
    gpu_memory_free();

    // Finalize MPI
    MPI_Finalize();

    // Exit
    exit(bSuccess == 1 ? 0 : 1);

}

/*************************************************************************************************************
 * FUNCTION:        initialize_pebbles
 *
 * DESCRIPTION:     Randomly distribute the specified number of pebbles in the lake, giving each a random size
 *
 * ARGUMENTS:       aPebbleSizes -          Array representing the pebble sizes at every point in the lake (sparse)
 *                  nPebbles -              The number of pebbles to drop into the lake
 *                  nLakePointsOneAxis -    The number of points in the map of the lake (one axis)
 *
 * RETURNS:         None
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *************************************************************************************************************/
void initialize_pebbles(double *aPebbleSizes, int nPebbles, int nLakePointsOneAxis)
{

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing initialize_pebbles\n", nMyRank);
        fflush(stdout);
    }

    // Declare variables
    int i, j, k, nIndex;
    int nPebbleSize;

    // Initialize
    srand( time(NULL) );
    memset(aPebbleSizes, 0, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis);

    // Loop through all pebbles, distributing them randomly into the lake
    for( k = 0; k < nPebbles ; k++ )
    {
        i = rand() % (nLakePointsOneAxis - 4) + 2;
        j = rand() % (nLakePointsOneAxis - 4) + 2;
        nPebbleSize = rand() % MAX_PSZ;
        nIndex = j + i * nLakePointsOneAxis;
        aPebbleSizes[nIndex] = (double) nPebbleSize;
    }

}

/*************************************************************************************************************
 * FUNCTION:        get_pebble_impact
 *
 * DESCRIPTION:     Get the energy impact of a given pebble size on the lake based on time.
 *                  Impact decreases as time increases.
 *
 * ARGUMENTS:       nPebbleSize -   The size of a given pebble
 *                  nTime -         The amount of time that has elapsed in the simulation
 *
 * RETURNS:         (unnamed) -     The energy impact of the specified pebble size given the current time
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *************************************************************************************************************/
double get_pebble_impact(double nPebbleSize, double nTime)
{
  return -expf(-TSCALE * nTime) * nPebbleSize;
}

/*************************************************************************************************************
 * FUNCTION:        initialize_energy
 *
 * DESCRIPTION:     Initialize the energy levels in the lake based solely upon initial drops of pebbles
 *
 * ARGUMENTS:       aEnergy -               Array representing energy levels at every point in the lake
 *                  aPebbleSizes -          Array representing the pebble sizes at every point in the lake (sparse)
 *                  nLakePointsOneAxis -    The number of points in the map of the lake (one axis)
 *                  bFullMap -              1 for the entire lake, 0 for this task's slice of the lake
 *
 * RETURNS:         None
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *************************************************************************************************************/
void initialize_energy(double *aEnergy, double *aPebbleSizes, int nLakePointsOneAxis, int bFullMap)
{

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing initialize_energy\n", nMyRank);
        fflush(stdout);
    }

    // Declare variables
    int nY;
    int nX;
    int nIndex;
    int nMinY;
    int nMaxY;

    // Determine maximum y axis value
    if (bFullMap == 1) {
        // Iterate over the entire lake
        nMinY = 0;
        nMaxY = nLakePointsOneAxis;
    }
    else {
        // Iterate over this task's section of the lake (ignoring boundary exchange space)
        nMinY = 1;
        nMaxY = (nLakePointsOneAxis / nNumTasks) + 1;
    }

    // Loop through both axes of the lake, initializing energy levels at each point
    for(nY = nMinY; nY < nMaxY ; nY++)
    {
        for(nX = 0; nX < nLakePointsOneAxis ; nX++)
        {
            nIndex = nX + nY * nLakePointsOneAxis;
            aEnergy[nIndex] = get_pebble_impact(aPebbleSizes[nIndex], 0.0);
        }
    }
}

/*************************************************************************************************************
 * FUNCTION:        print_heatmap
 *
 * DESCRIPTION:     Print (to file) a heat map showing the energy levels in the lake,
 *                  or in some portion of the lake.
 *
 * ARGUMENTS:       sFilename -     The name of the heat map file to write
 *                  aEnergy -       Array representing energy levels at every point in the lake
 *                  bFullMap -      True (one) if the entire 2d map, false (zero) if a single tasks
                                    space (which includes a upper and lower bound - must exclude)
 *
 * RETURNS:         None
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *************************************************************************************************************/
void print_heatmap (const char *sFilename, double *aEnergy, int bFullMap) {

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing print_heatmap\n", nMyRank);
        fflush(stdout);
    }

    // Declare variables
    int i, ii, j, nIndex;
    int imax;
    int jmax;

    // Initialize variables
    if (bFullMap == 1) {
        i = 0;
        j = 0;
        imax = nLakePointsOneAxis;
        jmax = nLakePointsOneAxis;
    } else {
        // Task map has outer bounds included
        i = 1;
        j = 0;
        imax = nLakePointsOneAxis / nNumTasks;
        jmax = nLakePointsOneAxis;
    }

    // Open file
    FILE *fp = fopen(sFilename, "w");


    // Work your way up the vertical axis
    for(ii = 0; ii < imax; ii++, i++ )
    {
        // Work your way across the horizontal axis
        for( j = 0; j < jmax; j++ )
        {
            nIndex = j + i * nLakePointsOneAxis;
            // Print: x, y, energy level
            fprintf(fp, "%f %f %f\n", j*nLakePointSpacing, ii*nLakePointSpacing, aEnergy[nIndex]);
        }
    }

    // Close file
    fclose(fp);

}

/*************************************************************************************************************
 * FUNCTION:        report_buffer_double
 *
 * DESCRIPTION:     Print (to terminal) the values in a buffer of double values - used for debug
 *
 * ARGUMENTS:       aBuffer -       Pointer to the buffer
 *                  nNumValues -    The number of values to print
 *
 * RETURNS:         None
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *************************************************************************************************************/
void report_buffer_double (double *aBuffer, int nNumValues) {

    // Print debug info
    if (B_DEBUG >= 1) {
        printf("Task rank %d executing report_buffer_double\n", nMyRank);
        fflush(stdout);
    }

    // Declare variables
    int i;

    // Print
    for(i = 0; i < nNumValues; i++ ) {
        printf("%f\n", aBuffer[i]);
        fflush(stdout);
    }

}
