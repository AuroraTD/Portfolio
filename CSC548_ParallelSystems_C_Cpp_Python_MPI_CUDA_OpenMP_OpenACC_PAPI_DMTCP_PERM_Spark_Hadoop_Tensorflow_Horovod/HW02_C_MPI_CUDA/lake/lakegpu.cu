/*************************************************************************************************************
 * FILE:            lakegpu.cu
 *
 * AUTHORS:	        attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *
 * DESCRIPTION:	    Assist with modeling the surface of a lake,
 *                      where some pebbles have been thrown onto the surface.
 *                  The energy level at any point on the lake is influenced by
 *                      the energy level on that point in the past,
 *                      and by the current energy levels at neighboring points.
 *                  This program takes into account all 8 neighboring points,
 *                      and parallelizes the simulation by using EXACTLY ONE compute node,
 *                      using multiple GPU threads.
 *
 * TO RUN:          srun -N1 -n1 -p opteron -x c[53,101,102] --pty /bin/bash
 *                  make -f p3.Makefile lake
 *                  prun ./lake [lake size] [# pebbles] [duration of simulation in seconds] [# GPU threads]
 *************************************************************************************************************/

// INCLUDES
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

// DEFINES
#define __DEBUG
#define TSCALE  1.0
#define VSQR    0.1

/*************************************************************************************************************
 * FUNCTION:        kf
 *
 * DESCRIPTION:     Get the energy impact of a given pebble size on the lake based on time.
 *                  Impact decreases as time increases.
 *
 * ARGUMENTS:       nPebbleSize -   The size of a given pebble
 *                  nTime -         The amount of time that has elapsed in the simulation
 *
 * AUTHORS:         ssbehera    Subhendu S. Behera
 *************************************************************************************************************/
__device__ double kf(double nPebbleSize, double nTime)
{
  return -expf(-TSCALE * nTime) * nPebbleSize;
}

/*************************************************************************************************************
 * FUNCTION:        evolve
 *
 * DESCRIPTION:     Update the energy levels in the lake for every lake point.
 *                  Each lake point's new energy level depends upon
 *                  old energy levels and the energy levels of neighboring points.
 *                  This version uses 9 points (point of interest and 8 neighboring points).
 *
 * ARGUMENTS:       aDeviceEnergy -         Array representing new energy levels at every point in the lake
 *                  aDeviceEnergyStep0 -    Array representing current energy levels at every point in the lake
 *                  aDeviceEnergyStep1 -    Array representing old energy levels at every point in the lake
 *                  aDevicePebbleSizes -    Array representing the pebble sizes at every point in the lake (sparse)
 *                  nTime -                 The amount of time that has elapsed in the simulation
 *                  nLakePointsOneAxis -    The number of points in the map of the lake (one axis)
 *                  timeStep -              The amount of time between one simulation step and the next
 *                  nPointSpacing -         The spacing between two points on the lake
 *
 * RETURNS:         None
 *
 * AUTHOR:          ssbehera    Subhendu S. Behera
 *************************************************************************************************************/
__global__ void evolve(
    double *aDeviceEnergy,
    double *aDeviceEnergyStep0,
    double *aDeviceEnergyStep1,
    double *aDevicePebbleSizes,
    float nTime,
    int nLakePointsOneAxis,
    float timeStep,
    double nPointSpacing
)
{
	int idx, idy;
	int nIndex;

	/*
	 * calculate idx & idy.
	 */
	idx = threadIdx.x + blockIdx.x * blockDim.x;
	idy = threadIdx.y + blockIdx.y * blockDim.y;

	if (idx <= nLakePointsOneAxis - 1 && idy <= nLakePointsOneAxis - 1) {
		/*
		 * calculate energy only if you are inside the lake.
		 */
		nIndex = idx * nLakePointsOneAxis + idy;
		if (idx == 0 || idx == nLakePointsOneAxis - 1 || idy == 0 || idy == nLakePointsOneAxis - 1)
			aDeviceEnergy[nIndex] = 0.;
		else
			aDeviceEnergy[nIndex] = 2 * aDeviceEnergyStep1[nIndex] - aDeviceEnergyStep0[nIndex] +
									VSQR * (timeStep * timeStep) * ((aDeviceEnergyStep1[nIndex - 1] +
									aDeviceEnergyStep1[nIndex + 1] + aDeviceEnergyStep1[nIndex + nLakePointsOneAxis]
									+ aDeviceEnergyStep1[nIndex - nLakePointsOneAxis] + 0.25 *
									(aDeviceEnergyStep1[nIndex + nLakePointsOneAxis - 1] +
									 aDeviceEnergyStep1[nIndex + nLakePointsOneAxis + 1] +
									 aDeviceEnergyStep1[nIndex - nLakePointsOneAxis - 1] +
									 aDeviceEnergyStep1[nIndex - nLakePointsOneAxis + 1]) -
									5 * aDeviceEnergyStep1[nIndex]) /
									(nPointSpacing * nPointSpacing) + kf(aDevicePebbleSizes[nIndex], nTime));
	}

}

/*************************************************************************************************************
 * FUNCTION:        run_gpu
 *
 * DESCRIPTION:     Simulate the energy changes over time in the lake, on the GPU
 *
 * ARGUMENTS:       aEnergy -               Array representing energy levels at every point in the lake
 *                  aEnergyStep0 -          Array representing energy levels at every point in the lake at time 0
 *                  aEnergyStep1 -          Array representing energy levels at every point in the lake at time 1
 *                  aPebbleSizes -          Array representing the pebble sizes at every point in the lake (sparse)
 *                  nLakePointsOneAxis -    The number of points in the map of the lake (one axis)
 *                  nPointSpacing -         The spacing between two points on the lake
 *                  nFinishTime -           The duration of the simulation in seconds
 *                  nThreads -              The number of threads to be used per block
 *                                              for instance, with nthreads=8,
 *                                              and a domain of grid points (nLakePointsOneAxis=128 x 128),
 *                                              you will create (nLakePointsOneAxis/nthreads)x(nLakePointsOneAxis/nthreads) = (16 x 16) blocks,
 *                                              with (8 x 8) threads on each block.
 *
 * RETURNS:         None
 *
 * AUTHORS:         ssbehera    Subhendu S. Behera
 *************************************************************************************************************/
void run_gpu(
    double *aEnergy,
    double *aEnergyStep0,
    double *aEnergyStep1,
    double *aPebbleSizes,
    int nLakePointsOneAxis,
    double nPointSpacing,
    double nFinishTime,
    int nThreads
)
{
	hipEvent_t kstart, kstop;
	float ktime, timeStep, nTime = 0.0f;
	double *aDeviceEnergy, *aDeviceEnergyStep0, *aDeviceEnergyStep1, *aDevicePebbleSizes;
	double *aEnergyCurrent, *aEnergyOld;
	int nLakePointsTotal = nLakePointsOneAxis * nLakePointsOneAxis;
	int blockDimension = (nLakePointsOneAxis / nThreads) +
						 (nLakePointsOneAxis % nThreads != 0 ? 1 : 0);
	dim3 threadsPerBlock(nThreads, nThreads);
	dim3 noOfBlocks(blockDimension, blockDimension);

	/*
	 * allocate host memory
	 */
	aEnergyCurrent = (double *)malloc(sizeof(double) * nLakePointsTotal);
	aEnergyOld = (double *)malloc(sizeof(double) * nLakePointsTotal);

	/*
	 * copy the data of energy step 0, step 1 to current & old energy respectively.
	 */
	memcpy(aEnergyOld, aEnergyStep0, sizeof(double) * nLakePointsTotal);
	memcpy(aEnergyCurrent, aEnergyStep1, sizeof(double) * nLakePointsTotal);

	/*
	 * allocate memory on the device.
	 */
	hipMalloc((void **)&aDeviceEnergy, sizeof(double) * nLakePointsTotal);
	hipMalloc((void **)&aDeviceEnergyStep0, sizeof(double) * nLakePointsTotal);
	hipMalloc((void **)&aDeviceEnergyStep1, sizeof(double) * nLakePointsTotal);
	hipMalloc((void **)&aDevicePebbleSizes, sizeof(double) * nLakePointsTotal);

	/*
	 * setup the timers before copying the memory from host to device.
	 */

	hipSetDevice(0);
	hipEventCreate(&kstart);
	hipEventCreate(&kstop);

	/*
	 * Start recording time.
	 */
	hipEventRecord(kstart, 0);

	/*
	 * copy the pebblesize data only once to device memory.
	 */
	hipMemcpy(aDevicePebbleSizes, aPebbleSizes, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis,
			   hipMemcpyHostToDevice);

	for (timeStep = nPointSpacing / 2; nTime < nFinishTime; nTime += timeStep) {
		/*
		 * copy data
		 */
		hipMemcpy(aDeviceEnergyStep0, aEnergyOld, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis,
				   hipMemcpyHostToDevice);
		hipMemcpy(aDeviceEnergyStep1, aEnergyCurrent, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis,
				   hipMemcpyHostToDevice);

		/*
	 	 * Make the kernel call.
	 	 */
		evolve<<<noOfBlocks, threadsPerBlock>>>(aDeviceEnergy, aDeviceEnergyStep0,
												aDeviceEnergyStep1, aDevicePebbleSizes,
												nTime, nLakePointsOneAxis, timeStep, nPointSpacing);

		/*
		 * copy the current energy to old energy as cpu is free.
		 */
		memcpy(aEnergyOld, aEnergyCurrent, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis);

		/*
		 * copy the new energy to current energy directly from the device.
		 */
		hipMemcpy(aEnergyCurrent, aDeviceEnergy, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis,
				   hipMemcpyDeviceToHost);
	}

	memcpy(aEnergy, aEnergyCurrent, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis);
	
	/* Stop GPU computation timer */
	hipEventRecord(kstop, 0);
	hipEventSynchronize(kstop);
	hipEventElapsedTime(&ktime, kstart, kstop);
	printf("GPU computation: %f msec\n", ktime);

	/*
	 * Free the device & host memory.
	 */
	free(aEnergyCurrent);
	free(aEnergyOld);
	hipFree(aDeviceEnergy);
	hipFree(aDeviceEnergyStep0);
	hipFree(aDeviceEnergyStep1);
	hipFree(aDevicePebbleSizes);

	/* timer cleanup */
	hipEventDestroy(kstart);
	hipEventDestroy(kstop);
}
