/*************************************************************************************************************
 * FILE:            lakegpu_mpi.cu
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *
 * DESCRIPTION:     Assist with modeling the surface of a slice of a lake,
 *                      where some pebbles have been thrown onto the surface.
 *                  The energy level at any point on the lake is influenced by
 *                      the energy level on that point in the past,
 *                      and by the current energy levels at neighboring points.
 *                  This program takes into account all 8 neighboring points,
 *                      and parallelizes the simulation by using EXACTLY ONE compute node,
 *                      using multiple GPU threads.
 *
 * TO RUN:          srun -N4 -n4 -p opteron -x c[53,101,102] --pty /bin/bash
 *                  make -f p3.Makefile lake-mpi
 *                  prun ./lake [lake size] [# pebbles] [duration of simulation in seconds] [# GPU threads]
 *************************************************************************************************************/

// INCLUDES
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

// DEFINES
#define __DEBUG
#define TSCALE  1.0
#define VSQR    0.1

// Declare globals
double *aDeviceEnergy, *aDeviceEnergyStepOld, *aDeviceEnergyStepCurrent, *aDevicePebbleSizes;

/*************************************************************************************************************
 * FUNCTION:        kf
 *
 * DESCRIPTION:     Get the energy impact of a given pebble size on the lake based on time.
 *                  Impact decreases as time increases.
 *
 * ARGUMENTS:       nPebbleSize -   The size of a given pebble
 *                  nTime -         The amount of time that has elapsed in the simulation
 *
 * AUTHORS:         ssbehera    Subhendu S. Behera
 *************************************************************************************************************/
__device__ double kf(double nPebbleSize, double nTime)
{
  return -expf(-TSCALE * nTime) * nPebbleSize;
}

/*************************************************************************************************************
 * FUNCTION:        evolve
 *
 * DESCRIPTION:     Update the energy levels in this node's slice of the lake for every lake point therein.
 *                  Each lake point's new energy level depends upon
 *                  old energy levels and the energy levels of neighboring points.
 *                  This version uses 9 points (point of interest and 8 neighboring points).
 *                  This code runs on 1 thread and is responsible for updating the energy level of 1 lake point.
 *
 * ARGUMENTS:       aDeviceEnergy -             Array representing new energy levels at every point in the lake
 *                  aDeviceEnergyStepOld -      Array representing current energy levels at every point in the lake
 *                  aDeviceEnergyStepCurrent -  Array representing old energy levels at every point in the lake
 *                  aDevicePebbleSizes -        Array representing the pebble sizes at every point in the lake (sparse)
 *                  nTime -                     The amount of time that has elapsed in the simulation
 *                  nLakePointsOneAxis -                  Number of lake points in this node's slice of the lake, on the x axis
 *                  nPointsY -                  Number of lake points in this node's slice of the lake, on the y axis
 *                  nTimeStep -                  The amount of time between one simulation step and the next
 *                  nPointSpacing -             The spacing between two points on the lake
 *
 * RETURNS:         None
 *
 * AUTHOR:          ssbehera    Subhendu S. Behera
 *                  attiffan    Aurora T. Tiffany-Davis
 *************************************************************************************************************/
__global__ void evolve(
    double *aDeviceEnergy,
    double *aDeviceEnergyStepOld,
    double *aDeviceEnergyStepCurrent,
    double *aDevicePebbleSizes,
    double nTime,
    int nPointsX,
	int nPointsY,
    double nTimeStep,
    double nPointSpacing,
	int nMyRank,
	int nNumTaskPoints
)
{

    // Declare variables
	int nIndexInMemory;

	int idx, idy;

	// Calculate thread index
	idx = threadIdx.x + blockIdx.x * blockDim.x;
	idy = threadIdx.y + blockIdx.y * blockDim.y;

	int nLakePointsOneAxis = nPointsX;

	// If the thread is actually needed to help with the calculation
	if (idx <= nPointsX - 1 && idy <= nPointsY - 1) {
		nIndexInMemory = (idy + 1) * (nPointsX) + idx;

		if (idy == 0 && nMyRank == 0) {
			aDeviceEnergy[nIndexInMemory] = 0.;
		}
		else if (idy == nPointsY - 1 && nMyRank == 3)
			aDeviceEnergy[nIndexInMemory] = 0.;
		else if (idx == 0 || idx == nPointsX - 1)
			aDeviceEnergy[nIndexInMemory] = 0.;
		else {
        aDeviceEnergy[nIndexInMemory] =
            2 * aDeviceEnergyStepCurrent[nIndexInMemory] -
            aDeviceEnergyStepOld[nIndexInMemory] +
            VSQR * (nTimeStep * nTimeStep) * (
                (
                    aDeviceEnergyStepCurrent[nIndexInMemory - 1] +
                    aDeviceEnergyStepCurrent[nIndexInMemory + 1] +
                    aDeviceEnergyStepCurrent[nIndexInMemory + nLakePointsOneAxis] +
                    aDeviceEnergyStepCurrent[nIndexInMemory - nLakePointsOneAxis] +
                    0.25 * (
                        aDeviceEnergyStepCurrent[nIndexInMemory + nLakePointsOneAxis - 1] +
                        aDeviceEnergyStepCurrent[nIndexInMemory + nLakePointsOneAxis + 1] +
                        aDeviceEnergyStepCurrent[nIndexInMemory - nLakePointsOneAxis - 1] +
                        aDeviceEnergyStepCurrent[nIndexInMemory - nLakePointsOneAxis + 1]
                    ) -
                    5 * aDeviceEnergyStepCurrent[nIndexInMemory]
                ) /
                (nPointSpacing * nPointSpacing) +
                kf(aDevicePebbleSizes[nIndexInMemory], nTime)
            );

		}
	}

}

/*************************************************************************************************************
 * FUNCTION:        gpu_memory_setup
 *
 * DESCRIPTION:     Allocate memory used by the GPU, and copy over pebble sizes array
 *                      (which does not change during execution).
 *
 * ARGUMENTS:       nNumTaskPointsWithBoundaries -  The total number of lake points that this node cares about
 *                                                  Includes this node's own lake points,
 *                                                      plus boundaries above / north and below / south
 *                  aPebbleSizes -                  Array representing pebbles in this node's slice of the lake
 *
 * RETURNS:         None
 *
 * AUTHOR:          ssbehera    Subhendu S. Behera
 *************************************************************************************************************/
void gpu_memory_setup(int nNumTaskPointsWithBoundaries, double *aPebbleSizes)
{

	hipMalloc((void **)&aDeviceEnergy, sizeof(double) * nNumTaskPointsWithBoundaries);
	hipMalloc((void **)&aDeviceEnergyStepOld, sizeof(double) * nNumTaskPointsWithBoundaries);
	hipMalloc((void **)&aDeviceEnergyStepCurrent, sizeof(double) * nNumTaskPointsWithBoundaries);
	hipMalloc((void **)&aDevicePebbleSizes, sizeof(double) * nNumTaskPointsWithBoundaries);
}

/*************************************************************************************************************
 * FUNCTION:        gpu_memory_free
 *
 * DESCRIPTION:     Free memory allocated by the GPU
 *
 * ARGUMENTS:       None
 *
 * RETURNS:         None
 *
 * AUTHOR:          ssbehera    Subhendu S. Behera
 *************************************************************************************************************/
void gpu_memory_free(void)
{
	/*
	 * Free the device memory.
	 */
	hipFree(aDeviceEnergy);
	hipFree(aDeviceEnergyStepOld);
	hipFree(aDeviceEnergyStepCurrent);
	hipFree(aDevicePebbleSizes);
}

/*************************************************************************************************************
 * FUNCTION:        run_gpu
 *
 * DESCRIPTION:     Simulate the energy changes over time in this node's slice of the lake, on the GPU
 *
 * ARGUMENTS:       aEnergyStepOld -                Array representing energy levels at every point in the lake at the previous time step
 *                  aEnergyStepCurrent -            Array representing energy levels at every point in the lake at the current time step
 *                  nLakePointsOneAxis -            The number of points in the map of the lake (one axis)
 *                  nNumTaskPointsWithBoundaries -  The total number of lake points that this node cares about
 *                                                  Includes this node's own lake points,
 *                                                      plus boundaries above / north and below / south
 *                  nPointSpacing -                 The spacing between two points on the lake
 *                  nTime -                         The current time value within the lake simulation
 *                  nThreads -                      The number of threads to be used per axis of 2D block (if block is 2D)
 *                                                  The intention is to have the total number of threads
 *                                                  equal the total number of lake points that this node is responsible for
 *
 * RETURNS:         None
 *
 * AUTHORS:         ssbehera    Subhendu S. Behera
 *************************************************************************************************************/
void run_gpu(
    double *aEnergyStepOld,
    double *aEnergyStepCurrent,
    int nLakePointsOneAxis,
    int nNumTaskPointsWithBoundaries,
    double nPointSpacing,
    double nTime,
    int nThreads, int nMyRank,
	int nNumTasks,
	double *aPebbleSizes
)
{

    // Declare variables
    double nTimeStep;
    int nNumTaskPoints;
	int nPointsY;

	/* Calculate block dimensions
	 *  One drawback of our design decision to split up the lake in slices instead of quadrants
	 *  is that this could result in some unused threads
	 */
	int blockDimensionX = (nLakePointsOneAxis / nThreads);
	if (blockDimensionX == 0)
		blockDimensionX++;

	int blockDimensionY = nLakePointsOneAxis / nNumTasks / nThreads;
	if (blockDimensionY == 0)
		blockDimensionY++;

	dim3 threadsPerBlock(nThreads, nThreads);
	dim3 noOfBlocks(blockDimensionX, blockDimensionY);


    // Calculate time step
	nTimeStep = nPointSpacing / 2;
	
	nNumTaskPoints = nNumTaskPointsWithBoundaries - (2 * nLakePointsOneAxis);

	nPointsY = nLakePointsOneAxis / nNumTasks;


	/*
	 * copy data
	 */
	hipMemcpy(aDeviceEnergyStepOld, aEnergyStepOld, sizeof(double) * nNumTaskPointsWithBoundaries,
			   hipMemcpyHostToDevice);
	hipMemcpy(aDeviceEnergyStepCurrent, aEnergyStepCurrent, sizeof(double) * nNumTaskPointsWithBoundaries,
			   hipMemcpyHostToDevice);

 	hipMemcpy(aDevicePebbleSizes, aPebbleSizes, sizeof(double) * nNumTaskPointsWithBoundaries,
               hipMemcpyHostToDevice);


	/*
	 * Make the kernel call.
	 */
	//nPointsY = nNumTaskPoints / nLakePointsOneAxis;
	evolve<<<noOfBlocks, threadsPerBlock>>>(
        aDeviceEnergy,
        aDeviceEnergyStepOld,
        aDeviceEnergyStepCurrent,
        aDevicePebbleSizes,
        nTime,
        nLakePointsOneAxis,
		nPointsY,
        nTimeStep,
        nPointSpacing,
		nMyRank,
		nNumTaskPoints
    );

	/*
	 * copy the current energy to old energy as cpu is free.
	 */
	memcpy(aEnergyStepOld, aEnergyStepCurrent, sizeof(double) * nNumTaskPointsWithBoundaries);

	/*
	 * copy the new energy to current energy directly from the device.
	 */
	hipMemcpy(aEnergyStepCurrent + nLakePointsOneAxis, aDeviceEnergy + nLakePointsOneAxis, sizeof(double) * nNumTaskPoints,
			   hipMemcpyDeviceToHost);
}
