/*************************************************************************************************************
 * FILE:            p2.cu
 *
 * AUTHOR:          attiffan    Aurora T. Tiffany-Davis
 *
 * DESCRIPTION:     A CUDA program that calculates an approximate value for PI using Monte Carlo methods.
 *
 * TO RUN:          srun -N1 -n1 -p opteron --pty /bin/bash
 *					make -f p2.Makefile
 *                  ./p2 [number of iterations for Monte Carlo simulation]
 *************************************************************************************************************/

// INCLUDES

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

// DEFINES
#define N_NUM_BLOCKS            64
#define N_NUM_THREADS_PER_BLOCK 512

/*************************************************************************************************************
 * FUNCTION:        iterateMonteCarlo
 *
 * DESCRIPTION:     GPU kernel that iterates through part of a Monte Carlo simulation for approximating PI.
 *
 * ARGUMENTS:       nThreads -      Number of threads which are participating in simulation
 *                  nIterations -   Total number of iterations to run
 *                  aCounts -       Pointer to an array of "raindrop" counts
 *
 * AUTHOR:          attiffan        Aurora T. Tiffany-Davis
 *************************************************************************************************************/
__global__ void iterateMonteCarlo (int nThreads, int nIterations, int *aCounts) {

    // Declare variables
    int         i;
    int         nThreadID;
    int         nCount;
    double      nX;
    double      nY;
    double      nZ;
    hiprandState state;

    // Initialize variables
    nThreadID = threadIdx.x + blockIdx.x * blockDim.x;
    nCount = 0;

    // If this thread is meant to participate, proceed
    if (nThreadID < nThreads) {

        // Initialize random number generator
        hiprand_init(clock64(), nThreadID, 0, &state);

        // Iterate through part of a Monte Carlo simulation, counting how many "raindrops" land in circle
        for (i = nThreadID; i < nIterations; i += nThreads) {

            nX = (double) hiprand_uniform(&state);
            nY = (double) hiprand_uniform(&state);
            nZ = (nX * nX) + (nY * nY);
            if (nZ <= 1) {
                nCount++;
            }

        }

        // Put your count in the counts array by thread index
        aCounts[nThreadID] = nCount;

    }

}

/*************************************************************************************************************
 * FUNCTION:        main
 *
 * DESCRIPTION:     Calculate an approximate value for PI using Monte Carlo methods.
 *
 * ARGUMENTS:       0 -         (as always, name of program)
 *                  1 -         Number of iterations to run
 *
 * AUTHOR:          attiffan    Aurora T. Tiffany-Davis
 *************************************************************************************************************/
int main(int argc, char** argv) {

    // Declare variables (CPU)
    double nPI;
    int i;
    int nTotalCount;
    int nTotalThreads = N_NUM_BLOCKS * N_NUM_THREADS_PER_BLOCK;
    int nIterations;
    int aCounts_h[nTotalThreads];

    // Declare variables (GPU)
    int *aCounts_d;

    // Initialize variables
    nIterations = atoi(argv[1]);
    nTotalCount = 0;

    // Allocate memory (GPU)
    hipMalloc((void **) &aCounts_d, sizeof(int) * nTotalThreads);

    // Run Monte Carlo simulation on GPU
    iterateMonteCarlo <<< N_NUM_BLOCKS, N_NUM_THREADS_PER_BLOCK >>> (nTotalThreads, nIterations, aCounts_d);

    // Copy back from GPU to CPU
    hipMemcpy(&aCounts_h, aCounts_d, sizeof(int) * nTotalThreads, hipMemcpyDeviceToHost);

    // Get total count of "raindrops" that fell within circle
    for (i = 0; i < nTotalThreads; i++) {
        nTotalCount += aCounts_h[i];
    }

    // Calculate approximate value of pi based on relationship between square and inscribed circle
    nPI = (double) nTotalCount / nIterations * 4;

    // Print results
    printf("# of trials= %d, estimate of pi is %.16f \n", nIterations, nPI);

    // Free memory (GPU)
    hipFree(aCounts_d);

}
