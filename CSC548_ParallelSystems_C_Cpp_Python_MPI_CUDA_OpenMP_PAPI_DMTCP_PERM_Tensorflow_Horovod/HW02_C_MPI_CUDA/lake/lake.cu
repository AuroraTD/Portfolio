/*************************************************************************************************************
 * FILE:            lake.cu
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *
 * DESCRIPTION:     Model the surface of a lake, where some pebbles have been thrown onto the surface.
 *                  The energy level at any point on the lake is influenced by
 *                      the energy level on that point in the past,
 *                      and by the current energy levels at neighboring points.
 *                  This program takes into account all 8 neighboring points,
 *                      and parallelizes the simulation by using EXACTLY ONE compute node,
 *                      optionally using multiple GPU threads.
 *                  Based upon the original lake.cu file, with added comments and more descriptive variable names.
 *
 * TO RUN:          srun -N1 -n1 -p opteron -x c[53,101,102] --pty /bin/bash
 *                  make -f p3.Makefile lake
 *                  prun ./lake [lake size] [# pebbles] [duration of simulation in seconds] [# GPU threads (optional)]
 *************************************************************************************************************/

// FUNCTION PROTOTYPES
int validate_inputs (int argc, char *argv[]);
int is_number (char sPossibleNumber[]);
void input_validation_error (const char *sMessage);
void initialize_energy (double *aEnergy, double *aPebbleSizes, int nLakePointsOneAxis);
void evolve (
    double *aEnergyNew,
    double *aEnergyCurrent,
    double *aEnergyOld,
    double *aPebbleSizes,
    int nLakePointsOneAxis,
    double nPointSpacing,
    double nTimeStep,
    double nTime
);
void evolve9pt (
    double *aEnergyNew,
    double *aEnergyCurrent,
    double *aEnergyOld,
    double *aPebbleSizes,
    int nLakePointsOneAxis,
    double nPointSpacing,
    double nTimeStep,
    double nTime
);
int update_time (double *nTime, double nTimeStep, double nFinishTime);
void print_heatmap (const char *sFilename, double *aEnergy, int nLakePointsOneAxis, double nPointSpacing);
void initialize_pebbles (double *aPebbleSizes, int nPebbles, int nLakePointsOneAxis);
void run_cpu (
    double *aEnergy,
    double *aEnergyStep0,
    double *aEnergyStep1,
    double *aPebbleSizes,
    int nLakePointsOneAxis,
    double nPointSpacing,
    double nFinishTime
);
extern void run_gpu (
    double *aEnergy,
    double *aEnergyStep0,
    double *aEnergyStep1,
    double *aPebbleSizes,
    int nLakePointsOneAxis,
    double nPointSpacing,
    double nFinishTime,
    int nThreads
);
double east (double *aEnergyCurrent, int nIndex);
double west (double *aEnergyCurrent, int nIndex);
double north (double *aEnergyCurrent, int nIndex, int nLakePointsOneAxis);
double northeast (double *aEnergyCurrent, int nIndex, int nLakePointsOneAxis);
double northwest (double *aEnergyCurrent, int nIndex, int nLakePointsOneAxis);
double south (double *aEnergyCurrent, int nIndex, int nLakePointsOneAxis);
double southeast (double *aEnergyCurrent, int nIndex, int nLakePointsOneAxis);
double southwest (double *aEnergyCurrent, int nIndex, int nLakePointsOneAxis);
double get_pebble_impact (double nPebbleSize, double nTime);

// INCLUDES

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stddef.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <ctype.h>

// DEFINES
#define _USE_MATH_DEFINES
#define XMIN    0.0
#define XMAX    1.0
#define YMIN    0.0
#define YMAX    1.0
#define MAX_PSZ 10
#define TSCALE  1.0
#define VSQR    0.1

// Hack to get around a problem I have in my IDE setup
#ifndef NULL
    #define NULL ((void *) 0)
#endif

/*************************************************************************************************************
 * FUNCTION:        main
 *
 * DESCRIPTION:     Model the surface of a lake, where some pebbles have been thrown onto the surface.
 *                  This work is parallelized by performing work on multiple GPU threads.
 *
 * ARGUMENTS:       0 -         Lake size (number of points along one axis)
 *                  1 -         Number of pebbles
 *                  2 -         Duration of the simulation in seconds
 *                  3 -         Number of GPU threads along one axis of each thread block
 *                              (optional - if omitted, run on CPU rather than on GPU)
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *************************************************************************************************************/
int main(int argc, char *argv[])
{

    // Check arguments
    int bSuccess = validate_inputs(argc, argv);
    if(bSuccess == 1) {

        // Declare variables

        int     nLakePointsOneAxis;
        int     nPebbles;
        double  nFinishTime;
        int     nThreads;
        int     nArea;

        double *aEnergyStep0, *aEnergyStep1;
        double *aEnergyCPU, *aEnergyGPU, *aPebbleSizes;
        double nPointSpacing;

        double nElapsedTimeCPU, nElapsedTimeGPU;
        struct timeval nStartTimeCPU, nEndTimeCPU, nStartTimeGPU, nEndTimeGPU;

        // Save arguments
        nLakePointsOneAxis    = atoi(argv[1]);
        nPebbles              = atoi(argv[2]);
        nFinishTime           = (double)atof(argv[3]);
        if (argc >= 5) {
            nThreads = atoi(argv[4]);
        }
        else {
            nThreads = -1;
        }
        nArea = nLakePointsOneAxis * nLakePointsOneAxis;

        // Allocate memory
        aEnergyStep0 = (double*)malloc(sizeof(double) * nArea);
        aEnergyStep1 = (double*)malloc(sizeof(double) * nArea);
        aPebbleSizes = (double*)malloc(sizeof(double) * nArea);
        aEnergyCPU = (double*)malloc(sizeof(double) * nArea);
        aEnergyGPU = (double*)malloc(sizeof(double) * nArea);

        // Let the user know what scenario we are running
        printf(
            "Running %s with %d threads, with (%d x %d) grid, with %d pebbles, until %f\n",
            argv[0],
            nThreads,
            nLakePointsOneAxis,
            nLakePointsOneAxis,
            nPebbles,
            nFinishTime
        );

        // Initialize
        nPointSpacing = (XMAX - XMIN)/nLakePointsOneAxis;
        initialize_pebbles(aPebbleSizes, nPebbles, nLakePointsOneAxis);
        initialize_energy(aEnergyStep0, aPebbleSizes, nLakePointsOneAxis);
        initialize_energy(aEnergyStep1, aPebbleSizes, nLakePointsOneAxis);

        // Print starting heat map to file
        print_heatmap("lake_i.dat", aEnergyStep0, nLakePointsOneAxis, nPointSpacing);

        // Run on CPU?
        if (nThreads < 0) {

            // Run simulation on CPU
            gettimeofday(&nStartTimeCPU, NULL);
            run_cpu(aEnergyCPU, aEnergyStep0, aEnergyStep1, aPebbleSizes, nLakePointsOneAxis, nPointSpacing, nFinishTime);
            gettimeofday(&nEndTimeCPU, NULL);

            // Report how long CPU took
            nElapsedTimeCPU = ((nEndTimeCPU.tv_sec + nEndTimeCPU.tv_usec * 1e-6)-(
                               nStartTimeCPU.tv_sec + nStartTimeCPU.tv_usec * 1e-6));
            printf("Entire job on CPU took %f seconds\n", nElapsedTimeCPU);

            // Print final heat map to file (use same name whether CPU or GPU)
            print_heatmap("lake_f.dat", aEnergyCPU, nLakePointsOneAxis, nPointSpacing);

        }

        // Run on GPU?
        else {

            // Run simulation on GPU
            gettimeofday(&nStartTimeGPU, NULL);
            run_gpu(aEnergyGPU, aEnergyStep0, aEnergyStep1, aPebbleSizes, nLakePointsOneAxis, nPointSpacing, nFinishTime, nThreads);
            gettimeofday(&nEndTimeGPU, NULL);

            // Report how long GPU took
            nElapsedTimeGPU = ((nEndTimeGPU.tv_sec + nEndTimeGPU.tv_usec * 1e-6)-(
                      nStartTimeGPU.tv_sec + nStartTimeGPU.tv_usec * 1e-6));
            printf("Entire job on GPU took %f seconds\n", nElapsedTimeGPU);

            // Print final heat map to file (use same name whether CPU or GPU)
            print_heatmap("lake_f.dat", aEnergyGPU, nLakePointsOneAxis, nPointSpacing);

        }

        // Free allocated memory
        free(aEnergyStep0);
        free(aEnergyStep1);
        free(aPebbleSizes);
        free(aEnergyCPU);
        free(aEnergyGPU);

    }

    // Return 0 if everything is okay
    return bSuccess == 1 ? 0 : 1;

}

/*************************************************************************************************************
 * FUNCTION:        validate_inputs
 *
 * DESCRIPTION:     Ensure the required command line arguments are present
 *
 * ARGUMENTS:       0 -         Lake size (number of points along one axis)
 *                  1 -         Number of pebbles
 *                  2 -         Duration of the simulation in seconds
 *                  3 -         Number of GPU threads along one axis of each thread block
 *                              OPTIONAL: "If GPU threads (nthreads) are specified from the command line,
 *                                          the GPU version should run, otherwise the CPU version"
 *
 * RETURNS:         bSuccess -  1 if inputs look okay, 0 otherwise
 *
 * AUTHOR:          attiffan    Aurora Tiffany-Davis
 *                  wpmoore2    Wade Moore
 *************************************************************************************************************/
int validate_inputs (int argc, char *argv[]) {

    // Declare variables
    int bSuccess;

    // Assume success until we know otherwise
    bSuccess = 1;

    // Check arguments
    if (argc < 4) {
        input_validation_error("Incorrect number of arguments provided");
        bSuccess = 0;
    }
    else if (
        is_number(argv[1]) != 1 ||
        atoi(argv[1]) <= 0
    ) {
        input_validation_error("Lake size must be a positive number");
        bSuccess = 0;
    }
    else if (
        is_number(argv[2]) != 1 ||
        atoi(argv[2]) <= 0
    ) {
        input_validation_error("Number of pebbles must be a positive number");
        bSuccess = 0;
    }
    else if (
        is_number(argv[3]) != 1 ||
        atoi(argv[3]) <= 0
    ) {
        input_validation_error("Duration of simulation must be a positive number");
        bSuccess = 0;
    }
    else if (
        argc >= 5 &&
        (
            is_number(argv[4]) != 1 ||
            atoi(argv[4]) <= 0
        )
    ) {
        input_validation_error("Number of GPU threads along one axis of each thread block must be a positive number");
        bSuccess = 0;
    }

    // Return
    return bSuccess;
}

/*************************************************************************************************************
 * FUNCTION:        is_number
 *
 * DESCRIPTION:     Checks to see if a char array represents a number
 *
 * ARGUMENTS:       sPossibleNumber - Something we hope actually represents a number
 *
 * RETURN:          1 if number, 0 otherwise
 *
 * SOURCE:          https://stackoverflow.com/questions/29248585/c-checking-command-line-argument-is-integer-or-not
 *
 * AUTHOR:          attiffan    Aurora T. Tiffany-Davis
 *************************************************************************************************************/
int is_number (char sPossibleNumber[]) {

    // Declare variable
    int bNumber = 1;
    int i = 0;

    // Checking for negative numbers
    if (sPossibleNumber[0] == '-') {
        i = 1;
    }
    for (; sPossibleNumber[i] != 0; i++)
    {
        //if (number[i] > '9' || number[i] < '0')
        if (!isdigit(sPossibleNumber[i]) && sPossibleNumber[i] != '.') {
            bNumber = 0;
            break;
        }
    }

    // Return
    return bNumber;

}

/*************************************************************************************************************
 * FUNCTION:        input_validation_error
 *
 * DESCRIPTION:     Complain about an input validation error
 *
 * ARGUMENTS:       sMessage -  A message about the error
 *
 * AUTHOR:          attiffan    Aurora Tiffany-Davis
 *                  wpmoore2    Wade Moore
 *************************************************************************************************************/
void input_validation_error (const char *sMessage) {

    printf("%s\n", sMessage);
    printf("\nUsage: prun ./lake-mpi [lake size] [# pebbles] [duration of simulation in seconds] [# GPU threads (optional)]\n");

}

/*************************************************************************************************************
 * FUNCTION:        run_cpu
 *
 * DESCRIPTION:     Simulate the energy changes over time in the lake, on the CPU
 *
 * ARGUMENTS:       aEnergy -               Array representing energy levels at every point in the lake
 *                  aEnergyStep0 -          Array representing energy levels at every point in the lake at time 0
 *                  aEnergyStep1 -          Array representing energy levels at every point in the lake at time 1
 *                  aPebbleSizes -          Array representing the pebble sizes at every point in the lake (sparse)
 *                  nLakePointsOneAxis -    The number of points in the map of the lake (one axis)
 *                  nPointSpacing -         The spacing between two points on the lake
 *                  nFinishTime -           The duration of the simulation in seconds
 *
 * RETURNS:         None
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *************************************************************************************************************/
void run_cpu(double *aEnergy, double *aEnergyStep0, double *aEnergyStep1, double *aPebbleSizes, int nLakePointsOneAxis, double nPointSpacing, double nFinishTime)
{

    // Declare variables
    double *aEnergyNew, *aEnergyCurrent, *aEnergyOld;
    double nTime, nTimeStep;

    // Allocate memory
    aEnergyNew = (double*)malloc(sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis);
    aEnergyCurrent = (double*)malloc(sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis);
    aEnergyOld = (double*)malloc(sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis);

    // Initialize
    memcpy(aEnergyOld, aEnergyStep0, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis);
    memcpy(aEnergyCurrent, aEnergyStep1, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis);
    nTime = 0.;
    nTimeStep = nPointSpacing / 2.;

    // Run simulation
    while(1)
    {
        #ifdef __FIVE_POINT
            evolve(aEnergyNew, aEnergyCurrent, aEnergyOld, aPebbleSizes, nLakePointsOneAxis, nPointSpacing, nTimeStep, nTime);
        #else
            evolve9pt(aEnergyNew, aEnergyCurrent, aEnergyOld, aPebbleSizes, nLakePointsOneAxis, nPointSpacing, nTimeStep, nTime);
        #endif
        memcpy(aEnergyOld, aEnergyCurrent, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis);
        memcpy(aEnergyCurrent, aEnergyNew, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis);
        if(!update_time(&nTime,nTimeStep,nFinishTime)) break;
    }

    // Finalize
    memcpy(aEnergy, aEnergyNew, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis);

}

/*************************************************************************************************************
 * FUNCTION:        initialize_pebbles
 *
 * DESCRIPTION:     Randomly distribute the specified number of pebbles in the lake, giving each a random size
 *
 * ARGUMENTS:       aPebbleSizes -          Array representing the pebble sizes at every point in the lake (sparse)
 *                  nPebbles -              The number of pebbles to drop into the lake
 *                  nLakePointsOneAxis -    The number of points in the map of the lake (one axis)
 *
 * RETURNS:         None
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *************************************************************************************************************/
void initialize_pebbles(double *aPebbleSizes, int nPebbles, int nLakePointsOneAxis)
{

    // Declare variables
    int i, j, k, nIndex;
    int nPebbleSize;

    // Initialize
    srand( time(NULL) );
    memset(aPebbleSizes, 0, sizeof(double) * nLakePointsOneAxis * nLakePointsOneAxis);

    // Loop through all pebbles, distributing them randomly into the lake
    for( k = 0; k < nPebbles ; k++ )
    {
        i = rand() % (nLakePointsOneAxis - 4) + 2;
        j = rand() % (nLakePointsOneAxis - 4) + 2;
        nPebbleSize = rand() % MAX_PSZ;
        nIndex = j + i * nLakePointsOneAxis;
        aPebbleSizes[nIndex] = (double) nPebbleSize;
    }

}

/*************************************************************************************************************
 * FUNCTION:        get_pebble_impact
 *
 * DESCRIPTION:     Get the energy impact of a given pebble size on the lake based on time.
 *                  Impact decreases as time increases.
 *
 * ARGUMENTS:       nPebbleSize -   The size of a given pebble
 *                  nTime -         The amount of time that has elapsed in the simulation
 *
 * RETURNS:         (unnamed) -     The energy impact of the specified pebble size given the current time
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *************************************************************************************************************/
double get_pebble_impact(double nPebbleSize, double nTime)
{
  return -expf(-TSCALE * nTime) * nPebbleSize;
}

/*************************************************************************************************************
 * FUNCTION:        update_time
 *
 * DESCRIPTION:     Update the current time in the simulation
 *
 * ARGUMENTS:       nTime -         The amount of time that has elapsed in the simulation
 *                  nTimeStep -     The amount of time between one simulation step and the next
 *                  nFinishTime -   The total intended duration of the simulation
 *
 * RETURNS:         bKeepGoing -    1 if we should keep going, 0 otherwise
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *************************************************************************************************************/
int update_time(double *nTime, double nTimeStep, double nFinishTime)
{

    // Declare variables
    int bKeepGoing;

    // Update the time counter
    if((*nTime) + nTimeStep > nFinishTime) {
        bKeepGoing = 0;
    }
    else {
        (*nTime) = (*nTime) + nTimeStep;
        bKeepGoing = 1;
    }

    // Return
    return bKeepGoing;

}

/*************************************************************************************************************
 * FUNCTION:        initialize_energy
 *
 * DESCRIPTION:     Initialize the energy levels in the lake based solely upon initial drops of pebbles
 *
 * ARGUMENTS:       aEnergy -               Array representing energy levels at every point in the lake
 *                  aPebbleSizes -          Array representing the pebble sizes at every point in the lake (sparse)
 *                  nLakePointsOneAxis -    The number of points in the map of the lake (one axis)
 *
 * RETURNS:         None
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *************************************************************************************************************/
void initialize_energy(double *aEnergy, double *aPebbleSizes, int nLakePointsOneAxis)
{

    // Declare variables
    int i, j, nIndex;

    // Loop through both axes of the lake, initializing energy levels at each point
    for(i = 0; i < nLakePointsOneAxis ; i++)
    {
        for(j = 0; j < nLakePointsOneAxis ; j++)
        {
            nIndex = j + i * nLakePointsOneAxis;
            aEnergy[nIndex] = get_pebble_impact(aPebbleSizes[nIndex], 0.0);
        }
    }

}

/*************************************************************************************************************
 * FUNCTION:        evolve
 *
 * DESCRIPTION:     Update the energy levels in the lake for every lake point.
 *                  Each lake point's new energy level depends upon
 *                  old energy levels and the energy levels of neighboring points.
 *                  This version uses 5 points (point of interest and 4 neighboring points).
 *
 * ARGUMENTS:       aEnergyNew -            Array representing new energy levels at every point in the lake
 *                  aEnergyCurrent -        Array representing current energy levels at every point in the lake
 *                  aEnergyOld -            Array representing old energy levels at every point in the lake
 *                  aPebbleSizes -          Array representing the pebble sizes at every point in the lake (sparse)
 *                  nLakePointsOneAxis -    The number of points in the map of the lake (one axis)
 *                  nPointSpacing -         The spacing between two points on the lake
 *                  nTimeStep -             The amount of time between one simulation step and the next
 *                  nTime -                 The amount of time that has elapsed in the simulation
 *
 * RETURNS:         None
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *************************************************************************************************************/
void evolve(double *aEnergyNew, double *aEnergyCurrent, double *aEnergyOld, double *aPebbleSizes, int nLakePointsOneAxis, double nPointSpacing, double nTimeStep, double nTime)
{

    // Declare variables
    int i, j, nIndex;

    // Loop through both axes of the lake, updating energy levels at each point
    for( i = 0; i < nLakePointsOneAxis; i++)
    {
        for( j = 0; j < nLakePointsOneAxis; j++)
        {
            nIndex = j + i * nLakePointsOneAxis;

            if( i == 0 || i == nLakePointsOneAxis - 1 || j == 0 || j == nLakePointsOneAxis - 1)
            {
                aEnergyNew[nIndex] = 0.;
            }
            else
            {
                aEnergyNew[nIndex] =
                    2*aEnergyCurrent[nIndex] -
                    aEnergyOld[nIndex] +
                    VSQR * (nTimeStep * nTimeStep) * (
                        (
                            west(aEnergyCurrent, nIndex) +
                            east(aEnergyCurrent, nIndex) +
                            north(aEnergyCurrent, nIndex, nLakePointsOneAxis) +
                            south(aEnergyCurrent, nIndex, nLakePointsOneAxis) -
                            4 * aEnergyCurrent[nIndex]
                        ) / (nPointSpacing * nPointSpacing) +
                        get_pebble_impact(aPebbleSizes[nIndex],nTime)
                    );
            }
        }
    }

}

/*************************************************************************************************************
 * FUNCTION:        evolve9pt
 *
 * DESCRIPTION:     Update the energy levels in the lake for every lake point.
 *                  Each lake point's new energy level depends upon
 *                  old energy levels and the energy levels of neighboring points.
 *                  This version uses 9 points (point of interest and 8 neighboring points).
 *
 * ARGUMENTS:       aEnergyNew -            Array representing new energy levels at every point in the lake
 *                  aEnergyCurrent -        Array representing current energy levels at every point in the lake
 *                  aEnergyOld -            Array representing old energy levels at every point in the lake
 *                  aPebbleSizes -          Array representing the pebble sizes at every point in the lake (sparse)
 *                  nLakePointsOneAxis -    The number of points in the map of the lake (one axis)
 *                  nPointSpacing -         The spacing between two points on the lake
 *                  nTimeStep -             The amount of time between one simulation step and the next
 *                  nTime -                 The amount of time that has elapsed in the simulation
 *
 * RETURNS:         None
 *
 * AUTHOR:          wpmoore2    Wade P. Moore
 *************************************************************************************************************/
void evolve9pt(double *aEnergyNew, double *aEnergyCurrent, double *aEnergyOld, double *aPebbleSizes, int nLakePointsOneAxis, double nPointSpacing, double nTimeStep, double nTime)
{

    // Declare variables
    int i, j, nIndex;

    // Loop through both axes of the lake, updating energy levels at each point
    for( i = 0; i < nLakePointsOneAxis; i++)
    {
        for( j = 0; j < nLakePointsOneAxis; j++)
        {
            nIndex = j + i * nLakePointsOneAxis;
            if( i == 0 || i == nLakePointsOneAxis - 1 || j == 0 || j == nLakePointsOneAxis - 1)
            {
                aEnergyNew[nIndex] = 0.;
            }
            else
            {
                aEnergyNew[nIndex] =
                    2*aEnergyCurrent[nIndex] -
                    aEnergyOld[nIndex] +
                    VSQR *(nTimeStep * nTimeStep) *(
                        (
                            west(aEnergyCurrent, nIndex) +
                            east(aEnergyCurrent, nIndex) +
                            north(aEnergyCurrent, nIndex, nLakePointsOneAxis) +
                            south(aEnergyCurrent, nIndex, nLakePointsOneAxis) +
                            + 0.25*(
                                northwest(aEnergyCurrent, nIndex, nLakePointsOneAxis) +
                                northeast(aEnergyCurrent, nIndex, nLakePointsOneAxis) +
                                southwest(aEnergyCurrent, nIndex, nLakePointsOneAxis) +
                                southeast(aEnergyCurrent, nIndex, nLakePointsOneAxis)
                            ) -
                            5 * aEnergyCurrent[nIndex]
                        ) / (nPointSpacing * nPointSpacing) +
                        get_pebble_impact(aPebbleSizes[nIndex],nTime)
                    );
            }
        }
    }

}

/*************************************************************************************************************
 * FUNCTION:        print_heatmap
 *
 * DESCRIPTION:     Print (to file) a heat map showing the energy levels in the lake
 *
 * ARGUMENTS:       sFilename -             The name of the heat map file to write
 *                  aEnergy -               Array representing energy levels at every point in the lake
 *                  nLakePointsOneAxis -    The number of points in the map of the lake (one axis)
 *                  nPointSpacing -         The spacing between two points on the lake
 *
 * RETURNS:         None
 *
 * AUTHORS:         attiffan    Aurora T. Tiffany-Davis
 *                  ssbehera    Subhendu S. Behera
 *                  wpmoore2    Wade P. Moore
 *************************************************************************************************************/
void print_heatmap(const char *sFilename, double *aEnergy, int nLakePointsOneAxis, double nPointSpacing)
{

    // Declare variables
    int i, j, nIndex;

    // Open file
    FILE *fp = fopen(sFilename, "w");

    // Loop through both axes of the lake, printing energy level at each point to file
    for( i = 0; i < nLakePointsOneAxis; i++ )
    {
        for( j = 0; j < nLakePointsOneAxis; j++ )
        {
            nIndex = j + i * nLakePointsOneAxis;
            fprintf(fp, "%f %f %f\n", i*nPointSpacing, j*nPointSpacing, aEnergy[nIndex]);
        }
    }

    // Close file
    fclose(fp);

}

/*************************************************************************************************************
 * FUNCTION:        (various)
 *
 * DESCRIPTION:     Get the current energy level of a neighboring point on the lake
 *
 * ARGUMENTS:       aEnergyCurrent -    Array representing current energy levels at every point in the lake
 *                  nIndex -            An index into the above array
 *
 * RETURNS:         (unnamed) -         The energy level at a neighboring point on the lake
 *
 * AUTHORS:         wpmoore2    Wade P. Moore
 *************************************************************************************************************/
double east(double *aEnergyCurrent, int nIndex) {
    return aEnergyCurrent[nIndex+1];
}

double west(double *aEnergyCurrent, int nIndex) {
    return aEnergyCurrent[nIndex-1];
}

double north(double *aEnergyCurrent, int nIndex, int nLakePointsOneAxis) {
    return aEnergyCurrent[nIndex + nLakePointsOneAxis];
}

double northwest(double *aEnergyCurrent, int nIndex, int nLakePointsOneAxis) {
    return aEnergyCurrent[nIndex + nLakePointsOneAxis - 1];
}

double northeast(double *aEnergyCurrent, int nIndex, int nLakePointsOneAxis) {
    return aEnergyCurrent[nIndex + nLakePointsOneAxis + 1];
}

double south(double *aEnergyCurrent, int nIndex, int nLakePointsOneAxis) {
    return aEnergyCurrent[nIndex - nLakePointsOneAxis];
}

double southwest(double *aEnergyCurrent, int nIndex, int nLakePointsOneAxis) {
    return aEnergyCurrent[nIndex - nLakePointsOneAxis - 1];
}

double southeast(double *aEnergyCurrent, int nIndex, int nLakePointsOneAxis) {
    return aEnergyCurrent[nIndex - nLakePointsOneAxis + 1];
}
